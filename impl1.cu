#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>

#include "utils.h"
#include "cuda_error_check.cuh"
#include "parse_graph.hpp"
#include "graph.h"

__global__ void pulling_kernel(std::vector<edge> * edges, std::vector<int>
    distances, int * hasChanged){

    //update me based on my neighbors. Toggle anyChange as needed.
    //offset will tell you who I am.
}

void puller(std::vector<edge> * edges, int blockSize, int blockNum){
    setTime();

    /*
     * Do all the things here!
     **/
    
    // sample way to access graph

    std::vector<edge> edges_data = *edges;

    for(int i = 0; i < edges->size(); i++){
      printf("Edge from %d to %d, with weight %d\n", edges_data[i].src,
          edges_data[i].dest, edges_data[i].weight);
    }

    std::cout << "Took " << getTime() << "ms.\n";
}
