#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>

#include "utils.h"
#include "cuda_error_check.cuh"
#include "graph.h"
#include "parse_graph.hpp"

__global__ void neighborHandling_kernel(std::vector<edge> * edges,
    std::vector<int> distances, int * hasChanged){

    //update me based on my neighbors. Toggle anyChange as needed.
    //Enqueue and dequeue me as needed.
    //Offset will tell you who I am.
}

void neighborHandler(std::vector<edge> * edges, int blockSize, int blockNum){
    setTime();

    /*
     * Do all the things here!
     **/

    std::cout << "Took " << getTime() << "ms.\n";
}
